﻿/*This code tests:
does system overhead will be too large when first calling cudaMallocAsync and explores why
*/
#include <hip/hip_runtime.h>

#include <iostream>


#define SIZE 64*1024*1024 //size of allocation

hipError_t test1() {

    int device = 0;// Choose which GPU to run on, change this on a multi-GPU system.
    std::cout << std::endl << "This code tests:  does system overhead will be too large when first calling hipMallocAsync and explores why." << std::endl << std::endl;
    std::cout << "Only cudaMallocAsync, not cudaFreeAsync---Allocate the same size each time" << std::endl << std::endl;
    hipError_t cudaStatus;
    int* dPtr[10];
    hipMemPoolProps poolProps = { };//set pool properties
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0;
    poolProps.location.type = hipMemLocationTypeDevice;
    poolProps.handleTypes = hipMemHandleTypeNone;
    hipMemPool_t memPool;
    hipStream_t stream;
    cudaStatus = hipMemPoolCreate(&memPool, &poolProps);//create explicit pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolCreate failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipDeviceSetMemPool(device, memPool);//set explicit pool as current pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSetMemPool failed!");
        return hipErrorInvalidValue;
    }

    unsigned long long int setVal = UINT64_MAX;//set threshold
    cudaStatus = hipMemPoolSetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &setVal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolSetAttribute failed!");
        return hipErrorInvalidValue;
    }
    float time = 0.0f;
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }


    for (int i = 0; i < 10; i++) {
        cudaStatus = hipEventRecord(start, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipEventRecord failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipMallocAsync((void**)&(dPtr[i]), SIZE, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMallocAsync failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipEventRecord(stop, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipEventRecord failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipEventSynchronize(stop);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipEventSynchronize failed!");
            return hipErrorInvalidValue;
        }
        hipEventElapsedTime(&time, start, stop);
        std::cout << i + 1 << " hipMallocAsync cost time is " << time << std::endl;
    }
    for (int i = 0; i < 10; i++) {
        cudaStatus = hipFreeAsync(dPtr[i], stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipFreeAsync failed!");
            return hipErrorInvalidValue;
        }
    }

    cudaStatus = hipStreamSynchronize(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamSynchronize failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolDestroy(memPool);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}

hipError_t test2() {

    int device = 0;// Choose which GPU to run on, change this on a multi-GPU system.
    std::cout << std::endl << "This code tests:  does system overhead will be too large when first calling hipMallocAsync and explores why." << std::endl << std::endl;
    std::cout << "cudaMallocAsync+cudaFreeAsync---Allocate the same size each time" << std::endl << std::endl;
    hipError_t cudaStatus;
    int* dPtr[10];
    hipMemPoolProps poolProps = { };//set pool properties
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0;
    poolProps.location.type = hipMemLocationTypeDevice;
    poolProps.handleTypes = hipMemHandleTypeNone;
    hipMemPool_t memPool;
    hipStream_t stream;
    cudaStatus = hipMemPoolCreate(&memPool, &poolProps);//create explicit pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolCreate failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipDeviceSetMemPool(device, memPool);//set explicit pool as current pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSetMemPool failed!");
        return hipErrorInvalidValue;
    }

    unsigned long long int setVal = UINT64_MAX;//set threshold
    cudaStatus = hipMemPoolSetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &setVal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolSetAttribute failed!");
        return hipErrorInvalidValue;
    }
    float time = 0.0f;
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }


    for (int i = 0; i < 10; i++) {
        cudaStatus = hipEventRecord(start, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipEventRecord failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipMallocAsync((void**)&(dPtr[i]), SIZE, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMallocAsync failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipEventRecord(stop, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipEventRecord failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipEventSynchronize(stop);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipEventSynchronize failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipFreeAsync(dPtr[i], stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipFreeAsync failed!");
            return hipErrorInvalidValue;
        }
        hipEventElapsedTime(&time, start, stop);
        std::cout << i + 1 << " hipMallocAsync cost time is " << time << std::endl;
    }

    cudaStatus = hipStreamSynchronize(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamSynchronize failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolDestroy(memPool);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}


hipError_t test3() {

    int device = 0;// Choose which GPU to run on, change this on a multi-GPU system.
    std::cout << std::endl << "This code tests:  does system overhead will be too large when first calling hipMallocAsync and explores why." << std::endl << std::endl;
    std::cout << "cudaMallocAsync+cudaFreeAsync---The allocation for each request is larger than before" << std::endl << std::endl;
    hipError_t cudaStatus;
    int* dPtr[10];
    hipMemPoolProps poolProps = { };//set pool properties
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0;
    poolProps.location.type = hipMemLocationTypeDevice;
    poolProps.handleTypes = hipMemHandleTypeNone;
    hipMemPool_t memPool;
    hipStream_t stream;
    cudaStatus = hipMemPoolCreate(&memPool, &poolProps);//create explicit pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolCreate failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipDeviceSetMemPool(device, memPool);//set explicit pool as current pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSetMemPool failed!");
        return hipErrorInvalidValue;
    }

    unsigned long long  int setVal = UINT64_MAX;//set threshold
    cudaStatus = hipMemPoolSetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &setVal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolSetAttribute failed!");
        return hipErrorInvalidValue;
    }
    float time = 0.0f;
    hipEvent_t start, stop;
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed!");
        return hipErrorInvalidValue;
    }


    for (int i = 0; i < 10; i++) {
        cudaStatus = hipEventRecord(start, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipEventRecord failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipMallocAsync((void**)&(dPtr[i]), (i+1)*SIZE, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMallocAsync failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipEventRecord(stop, stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipEventRecord failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipEventSynchronize(stop);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipEventSynchronize failed!");
            return hipErrorInvalidValue;
        }
        cudaStatus = hipFreeAsync(dPtr[i], stream);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipFreeAsync failed!");
            return hipErrorInvalidValue;
        }
        hipEventElapsedTime(&time, start, stop);
        std::cout << i + 1 << " hipMallocAsync cost time is " << time << std::endl;
    }

    cudaStatus = hipStreamSynchronize(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamSynchronize failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolDestroy(memPool);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;
}

int main()
{
    hipError_t cudaStatus;
    int device = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int driverVersion = 0;
    int deviceSupportsMemoryPools = 0;

    cudaStatus = hipDriverGetVersion(&driverVersion);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDriverGetVersion failed!");
        return 1;
    }
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
        (driverVersion % 100) / 10);

    if (driverVersion < 11040) {
        printf("Waiving execution as driver does not support Graph Memory Nodes\n");
        return 1;
    }

    cudaStatus = hipDeviceGetAttribute(&deviceSupportsMemoryPools,
        hipDeviceAttributeMemoryPoolsSupported, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetAttribute failed!");
        return 1;
    }
    if (!deviceSupportsMemoryPools) {
        printf("Waiving execution as device does not support Memory Pools\n");
        return 1;
    }
    else {
        printf("Running sample.\n");
    }

    cudaStatus = test1();//Only cudaMallocAsync, not cudaFreeAsync---Allocate the same size each time
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test1 failed!");
        return 1;
    }


 //   cudaStatus = test2();//cudaMallocAsync+cudaFreeAsync---Allocate the same size each time
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test2 failed!");
        return 1;
    }

//    cudaStatus = test3();//cudaMallocAsync+cudaFreeAsync---The allocation for each request is larger than before
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test3 failed!");
        return 1;
    }
    // cudaDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;

}
