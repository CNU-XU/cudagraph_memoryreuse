﻿/*This code tests:memory pool defragmentation*/
#include <hip/hip_runtime.h>
#include <iostream>


struct usageStatistics {
    cuuint64_t reserved;
    cuuint64_t reservedHigh;
    cuuint64_t used;
    cuuint64_t usedHigh;
};

hipError_t poolAttrGet(hipMemPool_t memPool, struct usageStatistics* statistics)
{
    std::cout << "-------MemPool Attribute-------" << std::endl;
    hipError_t cudaStatus;
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrReservedMemCurrent, &(statistics->reserved));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrReservedMemHigh, &(statistics->reservedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrUsedMemCurrent, &(statistics->used));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrUsedMemHigh, &(statistics->usedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "reserved is     : " << statistics->reserved << std::endl;
    std::cout << "reservedHigh is : " << statistics->reservedHigh << std::endl;
    std::cout << "used is         : " << statistics->used << std::endl;
    std::cout << "usedHigh is     : " << statistics->usedHigh << std::endl << std::endl;
    return hipSuccess;
}

hipError_t free1_3() {
    hipError_t cudaStatus;
    int device = 0; // Choose which GPU to run on, change this on a multi-GPU system.
    struct usageStatistics statistics = { 0,0,0,0 };

    hipMemPoolProps poolProps = { };//create explicit pool
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0;
    poolProps.location.type = hipMemLocationTypeDevice;
    poolProps.handleTypes = hipMemHandleTypeNone;
    hipMemPool_t memPool;
    hipStream_t stream;//create stream
    cudaStatus = hipMemPoolCreate(&memPool, &poolProps);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceSetMemPool(device, memPool);//set explicit pool as current pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSetMemPool failed!");
        return hipErrorInvalidValue;
    }
    unsigned int setVal = 1 << 30;//set threshold
    cudaStatus = hipMemPoolSetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &setVal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolSetAttribute failed!");
        return hipErrorInvalidValue;
    }
    int* d_a = NULL;
    int* d_b = NULL;
    int* d_c = NULL;
    int* d_d = NULL;
    int* d_f = NULL;
    int* d_g = NULL;

    cudaStatus = hipMallocAsync((void**)&d_a, 128 * 1024 * 1024, stream);//alloc128M
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipFreeAsync(d_a, stream);//free128M
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_b, 32 * 1024 * 1024, stream);//1alloc32M
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_c, 32 * 1024 * 1024, stream);//2alloc32M
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_d, 32 * 1024 * 1024, stream);//3alloc32M
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_f, 32 * 1024 * 1024, stream);//4alloc32M
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipFreeAsync(d_b, stream);//free1
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipFreeAsync(d_d, stream);//free3
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "--------------------this is free1_3---------------------" << std::endl;
    std::cout << "before alloc64" << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_g, 64 * 1024 * 1024, stream);//alloc64M
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "after alloc64" << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipFreeAsync(d_c, stream);//free2
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipFreeAsync(d_f, stream);//free4
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaFreeAsyncfailed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipMemPoolDestroy(memPool);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;

}

hipError_t free2_3() {
    hipError_t cudaStatus;
    int device = 0; // Choose which GPU to run on, change this on a multi-GPU system.
    struct usageStatistics statistics = { 0,0,0,0 };

    hipMemPoolProps poolProps = { };//create explicit pool
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0;
    poolProps.location.type = hipMemLocationTypeDevice;
    poolProps.handleTypes = hipMemHandleTypeNone;
    hipMemPool_t memPool;
    hipStream_t stream;//create stream
    cudaStatus = hipMemPoolCreate(&memPool, &poolProps);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceSetMemPool(device, memPool);//set explicit pool as current pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSetMemPool failed!");
        return hipErrorInvalidValue;
    }
    unsigned int setVal = 1 << 30;//set threshold
    cudaStatus = hipMemPoolSetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &setVal);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolSetAttribute failed!");
        return hipErrorInvalidValue;
    }
    int* d_a = NULL;
    int* d_b = NULL;
    int* d_c = NULL;
    int* d_d = NULL;
    int* d_f = NULL;
    int* d_g = NULL;

    cudaStatus = hipMallocAsync((void**)&d_a, 128 * 1024 * 1024, stream);//alloc128M
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipFreeAsync(d_a, stream);//free128M
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_b, 32 * 1024 * 1024, stream);//1alloc32M
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_c, 32 * 1024 * 1024, stream);//2alloc32M
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_d, 32 * 1024 * 1024, stream);//3alloc32M
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_f, 32 * 1024 * 1024, stream);//4alloc32M
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipFreeAsync(d_c, stream);//free2
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipFreeAsync(d_d, stream);//free3
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "--------------------this is free2_3---------------------" << std::endl;
    std::cout << "before alloc64" << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMallocAsync((void**)&d_g, 64 * 1024 * 1024, stream);//alloc64M
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "after alloc64" << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipFreeAsync(d_b, stream);//free1
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipFreeAsync(d_f, stream);//free4
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaFreeAsyncfailed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipMemPoolDestroy(memPool);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return hipSuccess;

}

int main()
{
    hipError_t cudaStatus;
    int device = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int driverVersion = 0;
    int deviceSupportsMemoryPools = 0;

    cudaStatus = hipDriverGetVersion(&driverVersion);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDriverGetVersion failed!");
        return 1;
    }
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
        (driverVersion % 100) / 10);

    if (driverVersion < 11040) {
        printf("Waiving execution as driver does not support Graph Memory Nodes\n");
        return 1;
    }

    cudaStatus = hipDeviceGetAttribute(&deviceSupportsMemoryPools,
        hipDeviceAttributeMemoryPoolsSupported, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetAttribute failed!");
        return 1;
    }
    if (!deviceSupportsMemoryPools) {
        printf("Waiving execution as device does not support Memory Pools\n");
        return 1;
    }
    else {
        printf("Running sample.\n");
    }
    std::cout << std::endl << "This code tests:memory pool defragmentation" << std::endl << std::endl;
    cudaStatus = free1_3();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "free1_3 failed!");
        return 1;
    }


   cudaStatus = free2_3();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "free2_3 failed!");
        return 1;
    }
    // cudaDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;

}

