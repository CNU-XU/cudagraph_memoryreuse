﻿/*This code tests:Does the implicit synchronization API free the memory pool*/
#include <hip/hip_runtime.h>

#include <iostream>

struct usageStatistics {
    cuuint64_t reserved;
    cuuint64_t reservedHigh;
    cuuint64_t used;
    cuuint64_t usedHigh;
};
hipError_t poolAttrGet(hipMemPool_t memPool, struct usageStatistics* statistics)
{
    std::cout << "-------MemPool Attribute-------" << std::endl;
    hipError_t cudaStatus;
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrReservedMemCurrent, &(statistics->reserved));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrReservedMemHigh, &(statistics->reservedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrUsedMemCurrent, &(statistics->used));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipMemPoolGetAttribute(memPool, hipMemPoolAttrUsedMemHigh, &(statistics->usedHigh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolGetAttribute failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "reserved is     : " << statistics->reserved << std::endl;
    std::cout << "reservedHigh is : " << statistics->reservedHigh << std::endl;
    std::cout << "used is         : " << statistics->used << std::endl;
    std::cout << "usedHigh is     : " << statistics->usedHigh << std::endl << std::endl;
    return hipSuccess;
}

hipError_t test1() {
    std::cout << std::endl << "This code tests:Does the implicit synchronization API free the memory pool" << std::endl << std::endl;
    int device = 0;
    hipError_t cudaStatus;
    int* bn = (int*)malloc(1 << 30);
    int* cn = 0ULL;
    int* dn = 0ULL;
    struct usageStatistics statistics = { 0,0,0,0 };
    cudaStatus = hipMalloc((void**)&cn, 1<<30);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return hipErrorInvalidValue;
    }
    hipMemPoolProps poolProps = { };//set pool properties
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0;
    poolProps.location.type = hipMemLocationTypeDevice;
    poolProps.handleTypes = hipMemHandleTypeNone;
    hipMemPool_t memPool;
    hipStream_t stream;
    cudaStatus = hipMemPoolCreate(&memPool, &poolProps);//create explicit pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolCreate failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipDeviceSetMemPool(device, memPool);//set explicit pool as current pool
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSetMemPool failed!");
        return hipErrorInvalidValue;
    }


    cudaStatus = hipMallocAsync((void**)&dn, 1 << 30, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocAsync failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamSynchronize(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamSynchronize failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "after mallocasync " << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }


 
    cudaStatus = hipFreeAsync(dn, stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFreeAsync failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "after freeasync" << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }


    cudaStatus = hipMemcpy(bn, cn, 1 << 30, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "after hipMemcpy" << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }


    cudaStatus = hipFree(cn);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFree failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "after hipFree" << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }

    cudaStatus = hipStreamSynchronize(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamSynchronize failed!");
        return hipErrorInvalidValue;
    }
    std::cout << "after streamsync" << std::endl;
    cudaStatus = poolAttrGet(memPool, &statistics);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "poolAttrGet failed!");
        return hipErrorInvalidValue;
    }
     
    cudaStatus = hipMemPoolDestroy(memPool);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemPoolDestroy failed!");
        return hipErrorInvalidValue;
    }
    cudaStatus = hipStreamDestroy(stream);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipStreamDestroy failed!");
        return hipErrorInvalidValue;
    }
    return cudaStatus;
}



int main()
{
    hipError_t cudaStatus;
    int device = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    int driverVersion = 0;
    int deviceSupportsMemoryPools = 0;

    cudaStatus = hipDriverGetVersion(&driverVersion);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDriverGetVersion failed!");
        return 1;
    }
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
        (driverVersion % 100) / 10);

    if (driverVersion < 11040) {
        printf("Waiving execution as driver does not support Graph Memory Nodes\n");
        return 1;
    }

    cudaStatus = hipDeviceGetAttribute(&deviceSupportsMemoryPools,
        hipDeviceAttributeMemoryPoolsSupported, device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceGetAttribute failed!");
        return 1;
    }
    if (!deviceSupportsMemoryPools) {
        printf("Waiving execution as device does not support Memory Pools\n");
        return 1;
    }
    else {
        printf("Running sample.\n");
    }

    cudaStatus = test1();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "test1 failed!");
        return 1;
    }


    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}


